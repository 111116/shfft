#include "hip/hip_runtime.h"
#include <hipfft/hipfft.h>
#include "shorder.hpp"
#include "select_size.hpp"

// DFT size: N*N
constexpr int N = select_size(n);

// convert from n*n SH vector to coefficients of Fourier Series
// placed at lower-most corner in the N*N array
__global__ void cu_sh2fs(float* SH, hipfftComplex* FS)
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    const int shbase = i*n*n;
    const int fsbase = i*N*N;
    // copy to register
    float SHreg[n*n];
    hipfftComplex FSreg[N*N];
    memcpy(SHreg, SH+shbase, n*n*sizeof(float));
    memset(FSreg, 0, N*N*sizeof(hipfftComplex));
    // execute
    #include "generated/sh2fs.cu"
    // copy back to global memory
   	for (int j=0; j<N*N; ++j)
   		FS[j+i*N*N] = FSreg[j];
}

// convert from coefficients of Fourier Series to SH vector
__global__ void cu_fs2sh(hipfftComplex* FS, float* SH)
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    const int shbase = i*n*n;
    const int fsbase = i*N*N;
    // copy to register
    float SHreg[n*n];
    hipfftComplex FSreg[N*N];
    memset(SHreg, 0, n*n*sizeof(float));
   	for (int j=0; j<N*N; ++j)
   		FSreg[j] = FS[j+i*N*N];
    // execute
    #include "generated/fs2sh.cu"
    // copy back to global memory
    memcpy(SH+shbase, SHreg, n*n*sizeof(float));
}

// element-wise multiplication B_i *= A_i
__global__ void multiply(hipfftComplex* A, hipfftComplex* B)
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
	float x = A[i].x * B[i].x - A[i].y * B[i].y;
	float y = A[i].y * B[i].x + A[i].x * B[i].y;
	B[i].x = x;
	B[i].y = y;
}

// A, B, C are pointers to SH coefficients in device memory
// layout: SH_0 [ at(0,0), at(1,-1), at(1,0), ... ], SH_1, ...
void shprod_many(float* A, float* B, float* C)
{
	const int blocksize = 32;
	assert(num%blocksize == 0);
	// mem alloc
	hipfftComplex *pool0, *pool1, *pool2;
	hipMalloc((void**)&pool0, sizeof(hipfftComplex)*N*N*num);
	hipMalloc((void**)&pool1, sizeof(hipfftComplex)*N*N*num);
	hipMalloc((void**)&pool2, sizeof(hipfftComplex)*N*N*num);
	// plan DFT
	hipfftHandle plan;
	int sizes[2] = {N,N};
	hipfftPlanMany(&plan, 2, sizes, NULL, 1, N*N, NULL, 1, N*N, HIPFFT_C2C, num);
    console.time("exclude_planning " + std::to_string(num));
	cu_sh2fs<<<num/blocksize, blocksize>>>(A, pool0);
    cu_sh2fs<<<num/blocksize, blocksize>>>(B, pool1);
	hipDeviceSynchronize();
    // DFT on A & B
    // console.time("fftexec " + std::to_string(num));
    hipfftExecC2C(plan, pool1, pool2, HIPFFT_FORWARD);
	hipfftExecC2C(plan, pool0, pool1, HIPFFT_FORWARD);
	// element-wise multiply
	multiply<<<num*N*N/blocksize, blocksize>>>(pool1, pool2);
	// IDFT & convert backs to SH
	hipfftExecC2C(plan, pool2, pool1, HIPFFT_BACKWARD);
	hipDeviceSynchronize();
    // console.timeEnd("fftexec " + std::to_string(num));
	cu_fs2sh<<<num/blocksize, blocksize>>>(pool1, C);
	// synchronize
	hipDeviceSynchronize();
    console.timeEnd("exclude_planning " + std::to_string(num));
}
